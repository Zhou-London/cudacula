#include "hip/hip_runtime.h"

#include <iostream>

#include "add_vector.h"

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

void Practices::add_vector(int N, int threadsPerBlock, const float* h_A,
                           const float* h_B, const float* h_C) {
    size_t size = N * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    std::cout << "hello\n";
}