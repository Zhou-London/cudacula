#include "hip/hip_runtime.h"
#include <iostream>
#include "mul_mat.h"

#define TILE_SIZE 16

__global__ void matMulTiled(float* A, float* B, float* C, int M, int N, int K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < M && t * TILE_SIZE + threadIdx.x < K)
            As[threadIdx.y][threadIdx.x] =
                A[row * K + t * TILE_SIZE + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && t * TILE_SIZE + threadIdx.y < K)
            Bs[threadIdx.y][threadIdx.x] =
                B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < N) C[row * N + col] = sum;
}

void Practices::mul_mat() {
    int M = 64, N = 64, K = 64;
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);

    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C = new float[M * N];

    for (int i = 0; i < M * K; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < K * N; ++i) h_B[i] = 2.0f;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    matMulTiled<<<grid, block>>>(d_A, d_B, d_C, M, N, K);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "C[0] = " << h_C[0] << std::endl;
    std::cout << "C[last] = " << h_C[M * N - 1] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}