
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloCUDA(){
    printf("Hello CUDA\n");
}

int main(){
    helloCUDA<<<1,5>>>();
    hipDeviceSynchronize();
    return 0;
}