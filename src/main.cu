#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>
#include <ranges>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 16;
    size_t size = N * sizeof(float);

    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    for (int i : std::ranges::views::iota(0, N)) {
        h_A[i] = i * 1.0f;
        h_B[i] = (N - i) * 1.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 8;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}